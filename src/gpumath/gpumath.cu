#include "hip/hip_runtime.h"
extern "C" {
    #include "gpumath.h"
}

#include "../math/units.h"
#include "../math/vector.h"

__global__ void gpuVecSub_kernel(Vector3* out, Vector3* a, Vector3* b, biguint n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id > n) return;

    out[id].values[0] = a[id].values[0] - b[id].values[0];
    out[id].values[1] = a[id].values[1] - b[id].values[1];
    out[id].values[2] = a[id].values[2] - b[id].values[2];
}

extern "C"
void gpuVecSub(Vector3* out, Vector3* a, Vector3* b, biguint n) {
    Vector3* gpuOut;
    Vector3* gpuA;
    Vector3* gpuB;
    hipMalloc(&gpuOut, sizeof(Vector3) * n);
    hipMalloc(&gpuA, sizeof(Vector3) * n);
    hipMalloc(&gpuB, sizeof(Vector3) * n);

    hipMemcpy(gpuA, a, sizeof(Vector3) * n, hipMemcpyHostToDevice);
    hipMemcpy(gpuB, b, sizeof(Vector3) * n, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksInGrid = ceil(float(n) / threadsPerBlock);
    gpuVecSub_kernel<<<blocksInGrid, threadsPerBlock>>>(gpuOut, gpuA, gpuB, n);

    hipMemcpy(out, gpuOut, sizeof(Vector3) * n, hipMemcpyDeviceToHost);
    hipFree(gpuOut);
    hipFree(gpuA);
    hipFree(gpuB);
}